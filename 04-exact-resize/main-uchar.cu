#include <iostream>
#include <vector>
#include "image.h"
#include "resize.h"
#include "error_check.h"

int main(int argc, char **argv) {

    if (argc != 3) {
        std::cout << "Expected 2 filenames to be provided: 1st is input, 2nd is output" << std::endl;
        return 1;
    }
    std::string filename_input(argv[1]);
    std::cout << "Loading " << filename_input << std::endl;
    
    unsigned char *input;
    int width;
    int height;
    int channels;
    const int scale = 8;
    
    int allocation_size_source = 0;
    auto input_image_allocation_function = [&allocation_size_source, &input](const int &allocation_size) -> void {
        if (allocation_size_source < allocation_size) {
            // If already has some allocation clear
            if (allocation_size_source > 0) {
                gpuErrchk(hipFree(&input));
            }
            gpuErrchk(hipMallocManaged(&input, allocation_size));
            allocation_size_source = allocation_size;
        }
    };
    auto get_ptr_function = [&input]() -> unsigned char ** {
        return &input;
    };
    read_jpeg_image_cu(filename_input, width, height, channels, input_image_allocation_function, get_ptr_function);

    std::cout << "Read image: " << width << " x " << height << " x " << channels << std::endl;

    int target_width = width / scale;
    int target_height = height / scale;
    const int allocation_size_target = width * height * channels;
    std::cout << "Target size: " << target_width << " x " << target_height << std::endl;
    
    // Separate image channels
    unsigned char *output;
    gpuErrchk(hipMallocManaged(&output, allocation_size_target));

    // Call resize kernel
    {
        resize_uchar(input, width, height, scale, output);
    }

    hipDeviceSynchronize();

    // Dump output
    std::string filename_output(argv[2]);
    std::cout << "Saving " << filename_output << std::endl;
    write_jpeg_image(filename_output, output, target_width, target_height, channels);

    hipFree(&input);
    hipFree(&output);

    std::cout << "Done!" << std::endl;
    return 0;
}
