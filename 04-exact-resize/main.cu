#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "image.h"
#include "resize.h"

int main(int argc, char **argv) {

    if (argc != 3) {
        std::cout << "Expected 2 filenames to be provided: 1st is input, 2nd is output" << std::endl;
        return 1;
    }
    std::string filename_input(argv[1]);
    std::cout << "Loading " << filename_input << std::endl;
    
    std::vector<unsigned char> image;
    int width;
    int height;
    int channels;
    const int scale = 2;
    
    read_jpeg_image(filename_input, image, width, height, channels);
    std::cout << "Read image: " << width << " x " << height << " x " << channels << std::endl;

    int target_width = width / scale;
    int target_height = height / scale;
    std::cout << "Target size: " << target_width << " x " << target_height << std::endl;
    
    // Separate image channels
    const int image_alloc_size_source = sizeof(float) * width * height;
    const int image_alloc_size_target = sizeof(float) * target_width * target_height;
    std::vector<float*> image_device(channels);
    std::vector<float*> result_device(channels);
    for (int channel = 0; channel < channels; channel++) {
        hipMallocManaged(&(image_device[channel]), image_alloc_size_source);
        hipMallocManaged(&(result_device[channel]), image_alloc_size_target);
    }
    
    // Copy image to GPU
    for (int row = 0; row < height; row++) {
        unsigned char *row_ptr_host = &image[row * width * channels];
        std::vector<float*> row_ptr_device;
        for (int channel = 0; channel < channels; channel++) {
            row_ptr_device.push_back(&(image_device[channel])[row * width]);
        }
        
        for (int col = 0; col <  width; col++)
            for (int channel = 0; channel < channels; channel++) {
                *(row_ptr_device[channel]) = static_cast<float>(*row_ptr_host);
                row_ptr_device[channel]++;
                row_ptr_host++;
            }
        
    }

    // Upload to GPU to prevent memory copy to be accounted profile
    for (int channel = 0; channel < channels; channel++) {
        hipMemPrefetchAsync(&(image_device[channel]), image_alloc_size_source, 0);
    }

    for (int channel = 0; channel < channels; channel++) {
        resize(image_device[channel], width, height, scale, result_device[channel]);
    }
    
    hipDeviceSynchronize();
    std::vector<unsigned char> image_result;
    image_result.resize(target_width * target_height * channels);

    std::cout << "Reading result" << std::endl;
    // Copy image from GPU
    for (int row = 0; row < target_height; row++) {
        unsigned char *row_ptr_host = &image_result[row * target_width * channels];
        std::vector<float*> row_ptr_device;
        for (int channel = 0; channel < channels; channel++) {
            row_ptr_device.push_back(&(result_device[channel])[row * target_width]);
        }
        
        for (int col = 0; col < target_width; col++)
            for (int channel = 0; channel < channels; channel++) {
                *row_ptr_host = 
                    static_cast<unsigned char>(
                        std::max(0.f, 
                        std::min(255.f, *(row_ptr_device[channel]))));
                row_ptr_device[channel]++;
                row_ptr_host++;
            }
        
    }
    
    std::string filename_output(argv[2]);
    std::cout << "Saving " << filename_output << std::endl;
    write_jpeg_image(filename_output, image_result, target_width, target_height, channels);
    
    for (int channel = 0; channel < channels; channel++) {
        hipFree(&(image_device[channel]));
        hipFree(&(result_device[channel]));
    }
    
    std::cout << "Done!" << std::endl;
    return 0;
}
